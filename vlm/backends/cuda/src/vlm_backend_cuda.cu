#include "vlm_backend_avx2.hpp"
#include "vlm_backend_cuda.hpp"

#include "simpletimer.hpp"

#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cstdio>

using namespace vlm;

#define CHECK_CUDA(call) \
    do { \
        hipError_t err = (call); \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA Error in %s at line %d: %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define CHECK_CUSOLVER(call) \
    do { \
        hipsolverStatus_t err = (call); \
        if (err != HIPSOLVER_STATUS_SUCCESS) { \
            fprintf(stderr, "cuSolver Error in %s at line %d: %d\n", \
                    __FILE__, __LINE__, err); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define CHECK_CUBLAS(call) \
    do { \
        hipblasStatus_t err = (call); \
        if (err != HIPBLAS_STATUS_SUCCESS) { \
            fprintf(stderr, "cuBLAS Error in %s at line %d: %d\n", \
                    __FILE__, __LINE__, err); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

void printCudaInfo() {
    // print out stats about the GPU in the machine.  Useful if
    // students want to know what GPU they are running on.

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    std::printf("----- CUDA Device information -----\n");
    std::printf("Found %d device(s)\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        std::printf("Device %d: %s\n", i, deviceProps.name);
        std::printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        std::printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        std::printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    std::printf("-----------------------------------\n");
}

// Singleton class to manage CUDA context (handle)
class CtxManager {
public:
    // This is how clients can access the single instance
    static CtxManager& getInstance() {
        static CtxManager instance;
        return instance;
    }

    void create() {
        // Create handlers
        CHECK_CUSOLVER(hipsolverDnCreate(&m_cusolver));
        CHECK_CUBLAS(hipblasCreate(&m_cublas));
        // Create CUDA stream
        CHECK_CUDA(hipStreamCreate(&m_stream));
        // Set handler streams
        CHECK_CUSOLVER(hipsolverSetStream(m_cusolver, m_stream));
        CHECK_CUBLAS(hipblasSetStream(m_cublas, m_stream));
    }

    void destroy() {
        CHECK_CUSOLVER(hipsolverDnDestroy(m_cusolver));
        CHECK_CUBLAS(hipblasDestroy(m_cublas));
        CHECK_CUDA(hipStreamDestroy(m_stream)); // destroy stream last
    }

    hipsolverHandle_t cusolver() {return(m_cusolver);}
    hipStream_t stream() {return(m_stream);}
    hipblasHandle_t cublas() {return(m_cublas);}

protected:
    hipsolverHandle_t m_cusolver = nullptr;
    hipStream_t m_stream = nullptr;
    hipblasHandle_t m_cublas = nullptr;

private:
    CtxManager() = default;
    ~CtxManager() = default;
};

BackendCUDA::BackendCUDA(Mesh& mesh, Data& data) : default_backend(mesh, data), Backend(mesh, data) {
    printCudaInfo();
    auto& ctx = CtxManager::getInstance();
    ctx.create();

    u64 n = (u64)mesh.nb_panels_wing();
    
    // Allocate device memory
    CHECK_CUDA(hipMalloc((void**)&d_lhs, n*n * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_rhs, n * sizeof(float)));
}

BackendCUDA::~BackendCUDA() {
    auto& ctx = CtxManager::getInstance();
    ctx.destroy();

    CHECK_CUDA(hipFree(d_lhs));
    CHECK_CUDA(hipFree(d_rhs));
}

// For the moment, cuda backend just falls back to AVX2

void BackendCUDA::reset() {
    default_backend.reset();
}

void BackendCUDA::compute_lhs() {
    default_backend.compute_lhs();
}

void BackendCUDA::compute_rhs() {
    default_backend.compute_rhs();
}

int CUDA_LU_solver(hipsolverHandle_t handle, float *d_A,
                   float *d_b, int n) {
    // All pointers are device pointers
    // A is column major 
    // Ax = b -> result is stored in b
    int bufferSize = 0;
    int *info = NULL;
    float *buffer = NULL;
    int *ipiv = NULL;  // pivoting sequence
    int h_info = 0;

    CHECK_CUSOLVER(hipsolverDnSgetrf_bufferSize(handle, n, n, (float *)d_A,
                                                n, &bufferSize));

    CHECK_CUDA(hipMalloc((void**)&info, sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&buffer, sizeof(float) * bufferSize));
    CHECK_CUDA(hipMalloc((void**)&ipiv, sizeof(int) * n));

    CHECK_CUDA(hipMemset(info, 0, sizeof(int)));

    CHECK_CUSOLVER(hipsolverDnSgetrf(handle, n, n, d_A, n, buffer, ipiv, info));
    CHECK_CUDA(
        hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost));

    if (0 != h_info) {
    fprintf(stderr, "Error: LU factorization failed\n");
    }

    CHECK_CUSOLVER(
        hipsolverDnSgetrs(handle, HIPBLAS_OP_N, n, 1, d_A, n, ipiv, d_b, n, info));
    CHECK_CUDA(hipDeviceSynchronize());

    if (info) {
    CHECK_CUDA(hipFree(info));
    }
    if (buffer) {
    CHECK_CUDA(hipFree(buffer));
    }
    if (ipiv) {
    CHECK_CUDA(hipFree(ipiv));
    }

    return 0;
}

void BackendCUDA::solve() {
    SimpleTimer timer("Solve");
    //default_backend.solve();
    auto& ctx = CtxManager::getInstance();
    u64 N = (u64)mesh.nb_panels_wing();

    // copy data to device
    CHECK_CUDA(hipMemcpy(d_lhs, default_backend.lhs.data(), N*N * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_rhs, default_backend.rhs.data(), N * sizeof(float), hipMemcpyHostToDevice));
    
    // Solve on device
    CUDA_LU_solver(ctx.cusolver(), d_lhs, d_rhs, N);

    // copy data back to host
    CHECK_CUDA(hipMemcpy(data.gamma.data(), d_rhs, N * sizeof(float), hipMemcpyDeviceToHost));
}

void BackendCUDA::compute_forces() {
    default_backend.compute_forces();
}

void BackendCUDA::compute_delta_gamma() {
    default_backend.compute_delta_gamma();
}
