#include "hip/hip_runtime.h"
#include "vlm_backend_cpu.hpp"
#include "vlm_backend_cuda.hpp"

#include "simpletimer.hpp"

#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "hip/hip_vector_types.h"

#include <cstdio>
#include <stdlib.h>
#include <hip/hip_vector_types.h>

using namespace vlm;

#define CHECK_CUDA(call) \
    do { \
        hipError_t err = (call); \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA Error in %s at line %d: %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define CHECK_CUSOLVER(call) \
    do { \
        hipsolverStatus_t err = (call); \
        if (err != HIPSOLVER_STATUS_SUCCESS) { \
            fprintf(stderr, "cuSolver Error in %s at line %d: %d\n", \
                    __FILE__, __LINE__, err); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define CHECK_CUBLAS(call) \
    do { \
        hipblasStatus_t err = (call); \
        if (err != HIPBLAS_STATUS_SUCCESS) { \
            fprintf(stderr, "cuBLAS Error in %s at line %d: %d\n", \
                    __FILE__, __LINE__, err); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

void printCudaInfo() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    std::printf("----- CUDA Device information -----\n");
    std::printf("Found %d device(s)\n", deviceCount);
    // Get CUDA Runtime version
    int cudaRuntimeVersion = 0;
    hipRuntimeGetVersion(&cudaRuntimeVersion);
    std::printf("CUDA Runtime: %d.%d\n", cudaRuntimeVersion / 1000, (cudaRuntimeVersion % 100) / 10);
    
    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        std::printf("Device %d: %s\n", i, deviceProps.name);
        std::printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        std::printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        std::printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    std::printf("-----------------------------------\n");
}

// Singleton class to manage CUDA context (handle)
class CtxManager {
public:
    // This is how clients can access the single instance
    static CtxManager& getInstance() {
        static CtxManager instance;
        return instance;
    }

    void create() {
        // Create handlers
        CHECK_CUSOLVER(hipsolverDnCreate(&m_cusolver));
        CHECK_CUBLAS(hipblasCreate(&m_cublas));
        // Create CUDA stream
        CHECK_CUDA(hipStreamCreate(&m_stream));
        // Set handler streams
        CHECK_CUSOLVER(hipsolverSetStream(m_cusolver, m_stream));
        CHECK_CUBLAS(hipblasSetStream(m_cublas, m_stream));
    }

    void destroy() {
        CHECK_CUSOLVER(hipsolverDnDestroy(m_cusolver));
        CHECK_CUBLAS(hipblasDestroy(m_cublas));
        CHECK_CUDA(hipStreamDestroy(m_stream)); // destroy stream last
    }

    hipsolverHandle_t cusolver() {return(m_cusolver);}
    hipStream_t stream() {return(m_stream);}
    hipblasHandle_t cublas() {return(m_cublas);}

protected:
    hipsolverHandle_t m_cusolver = nullptr;
    hipStream_t m_stream = nullptr;
    hipblasHandle_t m_cublas = nullptr;

private:
    CtxManager() = default;
    ~CtxManager() = default;
};

BackendCUDA::BackendCUDA(Mesh& mesh) : default_backend(mesh), Backend(mesh) {
    printCudaInfo();
    auto& ctx = CtxManager::getInstance();
    ctx.create();

    u64 n = mesh.nb_panels_wing();
    u64 npt = mesh.nb_panels_total();
    u64 nvt = mesh.nb_vertices_total();
    
    h_mesh.nb_panels = n;
    h_mesh.ns = mesh.ns;
    h_mesh.nc = mesh.nc;

    // Allocate device memory
    CHECK_CUDA(hipMalloc((void**)&d_lhs, n*n * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_rhs, n * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_gamma, n * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_delta_gamma, n * sizeof(float)));

    CHECK_CUDA(hipMalloc((void**)&h_mesh.v.x, nvt * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&h_mesh.v.y, nvt * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&h_mesh.v.z, nvt * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&h_mesh.colloc.x, npt * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&h_mesh.colloc.y, npt * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&h_mesh.colloc.z, npt * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&h_mesh.normal.x, npt * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&h_mesh.normal.y, npt * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&h_mesh.normal.z, npt * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_mesh, sizeof(MeshProxy)));

    // Prepare LU solver buffers
    int bufsize = 0;
    CHECK_CUSOLVER(hipsolverDnSgetrf_bufferSize(ctx.cusolver(), n, n, d_lhs, n, &bufsize));
    CHECK_CUDA(hipMalloc((void**)&d_solver_info, sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_solver_buffer, sizeof(float) * bufsize));
    CHECK_CUDA(hipMalloc((void**)&d_solver_ipiv, sizeof(int) * n));
}

BackendCUDA::~BackendCUDA() {
    auto& ctx = CtxManager::getInstance();
    ctx.destroy();

    CHECK_CUDA(hipFree(d_lhs));
    CHECK_CUDA(hipFree(d_rhs));
    CHECK_CUDA(hipFree(d_gamma));
    CHECK_CUDA(hipFree(d_delta_gamma));
    CHECK_CUDA(hipFree(h_mesh.v.x));
    CHECK_CUDA(hipFree(h_mesh.v.y));
    CHECK_CUDA(hipFree(h_mesh.v.z));
    CHECK_CUDA(hipFree(h_mesh.colloc.x));
    CHECK_CUDA(hipFree(h_mesh.colloc.y)); 
    CHECK_CUDA(hipFree(h_mesh.colloc.z));
    CHECK_CUDA(hipFree(h_mesh.normal.x));
    CHECK_CUDA(hipFree(h_mesh.normal.y));
    CHECK_CUDA(hipFree(h_mesh.normal.z));
    CHECK_CUDA(hipFree(d_mesh));
    CHECK_CUDA(hipFree(d_solver_info));
    CHECK_CUDA(hipFree(d_solver_buffer));
    CHECK_CUDA(hipFree(d_solver_ipiv));
}

// For the moment, cuda backend just falls back to cpu backend

void BackendCUDA::reset() {
    default_backend.reset();
    u64 n = mesh.nb_panels_wing();

    CHECK_CUDA(hipMemset(d_lhs, 0, n * n * sizeof(float)));
    CHECK_CUDA(hipMemset(d_rhs, 0, n * sizeof(float)));
    CHECK_CUDA(hipMemset(d_gamma, 0, n * sizeof(float)));
    CHECK_CUDA(hipDeviceSynchronize());
}

#define RCUT 1e-10f
#define RCUT2 1e-5f
#define PI_f 3.141593f
#define BlockSizeX 32
#define BlockSizeY 16

__device__ inline float3 kernel_biosavart(float3 colloc, const float3 vertex1, const float3 vertex2, const float sigma) {
    float3 r0 = vertex2 - vertex1;
    float3 r1 = colloc - vertex1;
    float3 r2 = colloc - vertex2;
    // Katz Plotkin, Low speed Aero | Eq 10.115
    float3 r1r2cross = cross(r1, r2);
    float r1_norm = length(r1);
    float r2_norm = length(r2);
    float square = length2(r1r2cross);
    
    if ((square<RCUT) || (r1_norm<RCUT2) || (r2_norm<RCUT2)) {
        float3 res = {0.0f, 0.0f, 0.0f};
        return res;
    }

    float smoother = sigma*sigma*length2(r0);

    float coeff = (dot(r0,r1)*r2_norm - dot(r0, r2)*r1_norm) / (4.0f*PI_f*sqrt(square*square + smoother*smoother)*r1_norm*r2_norm);
    return r1r2cross * coeff;
}

__device__ inline void kernel_symmetry(float3* inf, float3 colloc, const float3 vertex0, const float3 vertex1, const float sigma) {
    float3 induced_speed = kernel_biosavart(colloc, vertex0, vertex1, sigma);
    inf->x += induced_speed.x;
    inf->y += induced_speed.y;
    inf->z += induced_speed.z;
    colloc.y = -colloc.y; // wing symmetry
    float3 induced_speed_sym = kernel_biosavart(colloc, vertex0, vertex1, sigma);
    inf->x += induced_speed_sym.x;
    inf->y -= induced_speed_sym.y;
    inf->z += induced_speed_sym.z;
}

// start: starting linear index
// length: number of panels (columns) to process (from start to start+length)
// offset: offset between the linear index for the influenced panel and the influencing panel (used when influencing panel is part of the wake)
// Kernel achieves 32% of theoretical peak performance with 3.32 IPC and 85% of compute throughput
__global__ void kernel_influence_cuda(
    const MeshProxy* m,
    float* d_lhs,
    const uint64_t start, const uint64_t length, const uint64_t offset, const float sigma) {

    u64 j = blockIdx.y * blockDim.y + threadIdx.y;
    u64 i = blockIdx.x * blockDim.x + threadIdx.x;

    if (j >= length || i >= m->nb_panels) return;

    __shared__ float sharedCollocX[BlockSizeX];
    __shared__ float sharedCollocY[BlockSizeX];
    __shared__ float sharedCollocZ[BlockSizeX];
    __shared__ float sharedNormalX[BlockSizeX];
    __shared__ float sharedNormalY[BlockSizeX];
    __shared__ float sharedNormalZ[BlockSizeX];

    // Load memory along warp onto the shared memory
    if (threadIdx.y == 0) {
        // Load colloc and normal data into shared memory
        sharedCollocX[threadIdx.x] = m->colloc.x[i];
        sharedCollocY[threadIdx.x] = m->colloc.y[i];
        sharedCollocZ[threadIdx.x] = m->colloc.z[i];
        sharedNormalX[threadIdx.x] = m->normal.x[i];
        sharedNormalY[threadIdx.x] = m->normal.y[i];
        sharedNormalZ[threadIdx.x] = m->normal.z[i];
    }

    __syncthreads(); // Synchronize to ensure all shared mem data is loaded before proceeding

    float3 inf{0.0f, 0.0f, 0.0f};
    {
        const u64 v0 = (start + offset + j) + (start + offset + j) / m->ns;
        const u64 v1 = v0 + 1;
        const u64 v3 = v0 + m->ns + 1;
        const u64 v2 = v3 + 1;
        
        // Tried to put them in shared memory but got worse L1 hit rate. 
        const float3 vertex0{m->v.x[v0], m->v.y[v0], m->v.z[v0]};
        const float3 vertex1{m->v.x[v1], m->v.y[v1], m->v.z[v1]};
        const float3 vertex2{m->v.x[v2], m->v.y[v2], m->v.z[v2]};
        const float3 vertex3{m->v.x[v3], m->v.y[v3], m->v.z[v3]};

        // No bank conflicts as each thread reads a different index
        const float3 colloc = {sharedCollocX[threadIdx.x], sharedCollocY[threadIdx.x], sharedCollocZ[threadIdx.x]};

        kernel_symmetry(&inf, colloc, vertex0, vertex1, sigma);
        kernel_symmetry(&inf, colloc, vertex1, vertex2, sigma);
        kernel_symmetry(&inf, colloc, vertex2, vertex3, sigma);
        kernel_symmetry(&inf, colloc, vertex3, vertex0, sigma);
    }
    {
        const float3 normal = {sharedNormalX[threadIdx.x], sharedNormalY[threadIdx.x], sharedNormalZ[threadIdx.x]};
        d_lhs[(start + j) * m->nb_panels + i] += dot(inf, normal);
    }
}

constexpr u64 get_grid_size(u64 length, u64 block_size) {
    return (length + block_size - 1) / block_size;
}

void BackendCUDA::compute_lhs(const FlowData& flow) {
    SimpleTimer timer("LHS");
    // Copy the latest mesh that has been corrected for the aoa
    u64 npt = mesh.nb_panels_total();
    u64 nvt = mesh.nb_vertices_total();
    CHECK_CUDA(hipMemcpyAsync(h_mesh.v.x, mesh.v.x.data(), nvt * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpyAsync(h_mesh.v.y, mesh.v.y.data(), nvt * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpyAsync(h_mesh.v.z, mesh.v.z.data(), nvt * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpyAsync(h_mesh.colloc.x, mesh.colloc.x.data(), npt * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpyAsync(h_mesh.colloc.y, mesh.colloc.y.data(), npt * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpyAsync(h_mesh.colloc.z, mesh.colloc.z.data(), npt * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpyAsync(h_mesh.normal.x, mesh.normal.x.data(), npt * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpyAsync(h_mesh.normal.y, mesh.normal.y.data(), npt * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpyAsync(h_mesh.normal.z, mesh.normal.z.data(), npt * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipMemcpy(d_mesh, &h_mesh, sizeof(MeshProxy), hipMemcpyHostToDevice));

    dim3 block_size(BlockSizeX, BlockSizeY);
    dim3 grid_size(get_grid_size(mesh.nb_panels_wing(), block_size.x), get_grid_size((mesh.nc - 1) * mesh.ns, block_size.y));
    kernel_influence_cuda<<<grid_size, block_size>>>(d_mesh, d_lhs, 0, (mesh.nc - 1) * mesh.ns, 0, flow.sigma_vatistas);
    
    // hipError_t error = hipGetLastError();
    // if (error != hipSuccess) {
    //     fprintf(stderr, "CUDA Error after kernel launch: %s\n", hipGetErrorString(error));
    // }
    
    // CHECK_CUDA(hipDeviceSynchronize());

    dim3 grid_size2(get_grid_size(mesh.nb_panels_wing(), block_size.x), get_grid_size(mesh.ns, block_size.y));
    for (u64 offset = 0; offset < mesh.nw + 1; offset++) {
        kernel_influence_cuda<<<grid_size2, block_size>>>(d_mesh, d_lhs, (mesh.nc - 1) * mesh.ns, mesh.ns, offset*mesh.ns, flow.sigma_vatistas);
        // hipError_t error = hipGetLastError();
        // if (error != hipSuccess) {
        //     fprintf(stderr, "CUDA Error after kernel launch: %s\n", hipGetErrorString(error));
        // }
        CHECK_CUDA(hipDeviceSynchronize());
    }
}

void BackendCUDA::compute_rhs(const FlowData& flow) {
    default_backend.compute_rhs(flow);
}

void BackendCUDA::compute_rhs(const FlowData& flow, const std::vector<f32>& section_alphas) {
    default_backend.compute_rhs(flow, section_alphas);
}

void BackendCUDA::lu_factor() {
    SimpleTimer timer("Factor");
    int n = (int)mesh.nb_panels_wing();
    int h_info = 0;

    CHECK_CUSOLVER(hipsolverDnSgetrf(CtxManager::getInstance().cusolver(), n, n, d_lhs, n, d_solver_buffer, d_solver_ipiv, d_solver_info));
    CHECK_CUDA(hipMemcpy(&h_info, d_solver_info, sizeof(int), hipMemcpyDeviceToHost)); // sync
    if (h_info != 0) printf("Error: LU factorization failed\n");
};

void BackendCUDA::lu_solve() {
    SimpleTimer timer("Solve");
    //default_backend.solve();
    int n = (int)mesh.nb_panels_wing();
    int h_info = 0;

    // copy data to device (temporary)
    CHECK_CUDA(hipMemcpy(d_rhs, default_backend.rhs.data(), n * sizeof(float), hipMemcpyHostToDevice));
    
    // Solve on device
    CHECK_CUSOLVER(hipsolverDnSgetrs(CtxManager::getInstance().cusolver(), HIPBLAS_OP_N, n, 1, d_lhs, n, d_solver_ipiv, d_rhs, n, d_solver_info));
    CHECK_CUDA(hipMemcpy(&h_info, d_solver_info, sizeof(int), hipMemcpyDeviceToHost)); // sync
    if (h_info != 0) printf("Error: LU solve failed\n");

    // copy data back to host
    CHECK_CUDA(hipMemcpy(default_backend.gamma.data(), d_rhs, n * sizeof(float), hipMemcpyDeviceToHost));
}

f32 BackendCUDA::compute_coefficient_cl(
    const FlowData& flow,
    const f32 area,
    const u64 j,
    const u64 n) {
    return default_backend.compute_coefficient_cl(flow, area, j, n);
}

f32 BackendCUDA::compute_coefficient_cd(
    const FlowData& flow,
    const f32 area,
    const u64 j,
    const u64 n) {
    return default_backend.compute_coefficient_cd(flow, area, j, n);
}

linalg::alias::float3 BackendCUDA::compute_coefficient_cm(
    const FlowData& flow,
    const f32 area,
    const f32 chord,
    const u64 j,
    const u64 n) {
    return default_backend.compute_coefficient_cm(flow, area, chord, j, n);
}

void BackendCUDA::compute_delta_gamma() {
    default_backend.compute_delta_gamma();
}
