#include "hip/hip_runtime.h"
#include "vlm_backend_cpu.hpp"
#include "vlm_backend_cuda.hpp"

#include "simpletimer.hpp"

#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "hip/hip_vector_types.h"

#include <cstdio>
#include <stdlib.h>
#include <hip/hip_vector_types.h>

using namespace vlm;

#define CHECK_CUDA(call) \
    do { \
        hipError_t err = (call); \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA Error in %s at line %d: %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define CHECK_CUSOLVER(call) \
    do { \
        hipsolverStatus_t err = (call); \
        if (err != HIPSOLVER_STATUS_SUCCESS) { \
            fprintf(stderr, "cuSolver Error in %s at line %d: %d\n", \
                    __FILE__, __LINE__, err); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define CHECK_CUBLAS(call) \
    do { \
        hipblasStatus_t err = (call); \
        if (err != HIPBLAS_STATUS_SUCCESS) { \
            fprintf(stderr, "cuBLAS Error in %s at line %d: %d\n", \
                    __FILE__, __LINE__, err); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

void printCudaInfo() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    std::printf("----- CUDA Device information -----\n");
    std::printf("Found %d device(s)\n", deviceCount);
    // Get CUDA Runtime version
    int cudaRuntimeVersion = 0;
    hipRuntimeGetVersion(&cudaRuntimeVersion);
    std::printf("CUDA Runtime: %d.%d\n", cudaRuntimeVersion / 1000, (cudaRuntimeVersion % 100) / 10);
    
    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        std::printf("Device %d: %s\n", i, deviceProps.name);
        std::printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        std::printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        std::printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    std::printf("-----------------------------------\n");
}

// Singleton class to manage CUDA context (handle)
class CtxManager {
public:
    // This is how clients can access the single instance
    static CtxManager& getInstance() {
        static CtxManager instance;
        return instance;
    }

    void create() {
        // Create handlers
        CHECK_CUSOLVER(hipsolverDnCreate(&m_cusolver));
        CHECK_CUBLAS(hipblasCreate(&m_cublas));
        // Create CUDA stream
        CHECK_CUDA(hipStreamCreate(&m_stream));
        // Set handler streams
        CHECK_CUSOLVER(hipsolverSetStream(m_cusolver, m_stream));
        CHECK_CUBLAS(hipblasSetStream(m_cublas, m_stream));
    }

    void destroy() {
        CHECK_CUSOLVER(hipsolverDnDestroy(m_cusolver));
        CHECK_CUBLAS(hipblasDestroy(m_cublas));
        CHECK_CUDA(hipStreamDestroy(m_stream)); // destroy stream last
    }

    hipsolverHandle_t cusolver() {return(m_cusolver);}
    hipStream_t stream() {return(m_stream);}
    hipblasHandle_t cublas() {return(m_cublas);}

protected:
    hipsolverHandle_t m_cusolver = nullptr;
    hipStream_t m_stream = nullptr;
    hipblasHandle_t m_cublas = nullptr;

private:
    CtxManager() = default;
    ~CtxManager() = default;
};

BackendCUDA::BackendCUDA(Mesh& mesh) : default_backend(mesh), Backend(mesh) {
    printCudaInfo();
    auto& ctx = CtxManager::getInstance();
    ctx.create();

    u64 n = mesh.nb_panels_wing();
    u64 npt = mesh.nb_panels_total();
    u64 nvt = mesh.nb_vertices_total();
    
    // Allocate device memory
    CHECK_CUDA(hipMalloc((void**)&d_lhs, n*n * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_rhs, n * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_gamma, n * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_delta_gamma, n * sizeof(float)));

    h_mesh.nb_panels = n;
    h_mesh.ns = mesh.ns;
    h_mesh.nc = mesh.nc;
    CHECK_CUDA(hipMalloc((void**)&h_mesh.v.x, nvt * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&h_mesh.v.y, nvt * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&h_mesh.v.z, nvt * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&h_mesh.colloc.x, npt * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&h_mesh.colloc.y, npt * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&h_mesh.colloc.z, npt * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&h_mesh.normal.x, npt * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&h_mesh.normal.y, npt * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&h_mesh.normal.z, npt * sizeof(float)));

    CHECK_CUDA(hipMalloc((void**)&d_mesh, sizeof(MeshProxy)));      
    CHECK_CUDA(hipDeviceSynchronize());
}

BackendCUDA::~BackendCUDA() {
    auto& ctx = CtxManager::getInstance();
    ctx.destroy();

    CHECK_CUDA(hipFree(d_lhs));
    CHECK_CUDA(hipFree(d_rhs));
    CHECK_CUDA(hipFree(d_gamma));
    CHECK_CUDA(hipFree(d_delta_gamma));
    CHECK_CUDA(hipFree(h_mesh.v.x));
    CHECK_CUDA(hipFree(h_mesh.v.y));
    CHECK_CUDA(hipFree(h_mesh.v.z));
    CHECK_CUDA(hipFree(h_mesh.colloc.x));
    CHECK_CUDA(hipFree(h_mesh.colloc.y)); 
    CHECK_CUDA(hipFree(h_mesh.colloc.z));
    CHECK_CUDA(hipFree(h_mesh.normal.x));
    CHECK_CUDA(hipFree(h_mesh.normal.y));
    CHECK_CUDA(hipFree(h_mesh.normal.z));
    CHECK_CUDA(hipFree(d_mesh));
}

// For the moment, cuda backend just falls back to cpu backend

void BackendCUDA::reset() {
    default_backend.reset();
    u64 n = mesh.nb_panels_wing();

    CHECK_CUDA(hipMemset(d_lhs, 0, n * n * sizeof(float)));
    CHECK_CUDA(hipMemset(d_rhs, 0, n * sizeof(float)));
    CHECK_CUDA(hipMemset(d_gamma, 0, n * sizeof(float)));
    CHECK_CUDA(hipDeviceSynchronize());
}

#define RCUT 1e-10f
#define RCUT2 1e-5f

#define PI_f 3.141593f
#define BlockSizeX 16
#define BlockSizeY 16

__device__ inline float3 kernel_biosavart(float3 colloc, const float3 vertex1, const float3 vertex2, const float sigma) {
    float3 r0 = vertex2 - vertex1;
    float3 r1 = colloc - vertex1;
    float3 r2 = colloc - vertex2;
    // Katz Plotkin, Low speed Aero | Eq 10.115
    float3 r1r2cross = cross(r1, r2);
    float r1_norm = length(r1);
    float r2_norm = length(r2);
    float square = length2(r1r2cross);
    
    if ((square<RCUT) || (r1_norm<RCUT2) || (r2_norm<RCUT2)) {
        float3 res = {0.0f, 0.0f, 0.0f};
        return res;
    }

    float smoother = sigma*sigma*length2(r0);

    float coeff = (dot(r0,r1)*r2_norm - dot(r0, r2)*r1_norm) / (4.0f*PI_f*sqrt(square*square + smoother*smoother)*r1_norm*r2_norm);
    return r1r2cross * coeff;
}

__device__ inline void kernel_symmetry(float3* inf, float3 colloc, const float3 vertex0, const float3 vertex1, const float sigma) {
    float3 induced_speed = kernel_biosavart(colloc, vertex0, vertex1, sigma);
    inf->x += induced_speed.x;
    inf->y += induced_speed.y;
    inf->z += induced_speed.z;
    colloc.y = -colloc.y; // wing symmetry
    float3 induced_speed_sym = kernel_biosavart(colloc, vertex0, vertex1, sigma);
    inf->x += induced_speed_sym.x;
    inf->y -= induced_speed_sym.y;
    inf->z += induced_speed_sym.z;
}

// start: starting linear index
// length: number of panels (columns) to process (from start to start+length)
// offset: offset between the linear index for the influenced panel and the influencing panel (used when influencing panel is part of the wake)
__global__ void kernel_influence_cuda(
    const MeshProxy* m,
    float* d_lhs,
    const uint64_t start, const uint64_t length, const uint64_t offset, const float sigma) {

    u64 j = blockIdx.y * blockDim.y + threadIdx.y;
    u64 i = blockIdx.x * blockDim.x + threadIdx.x;

    if (j >= length || i >= m->nb_panels) return;

    const u64 v0 = (start + offset + j) + (start + offset + j) / m->ns;
    const u64 v1 = v0 + 1;
    const u64 v3 = v0 + m->ns + 1;
    const u64 v2 = v3 + 1;
    
    const float3 vertex0{m->v.x[v0], m->v.y[v0], m->v.z[v0]};
    const float3 vertex1{m->v.x[v1], m->v.y[v1], m->v.z[v1]};
    const float3 vertex2{m->v.x[v2], m->v.y[v2], m->v.z[v2]};
    const float3 vertex3{m->v.x[v3], m->v.y[v3], m->v.z[v3]};

    const float3 colloc = {m->colloc.x[i], m->colloc.y[i], m->colloc.z[i]};
    const float3 normal = {m->normal.x[i], m->normal.y[i], m->normal.z[i]};
    float3 inf{0.0f, 0.0f, 0.0f};
    kernel_symmetry(&inf, colloc, vertex0, vertex1, sigma);
    kernel_symmetry(&inf, colloc, vertex1, vertex2, sigma);
    kernel_symmetry(&inf, colloc, vertex2, vertex3, sigma);
    kernel_symmetry(&inf, colloc, vertex3, vertex0, sigma);
    d_lhs[(start + j) * m->nb_panels + i] += dot(inf, normal);
    // if (i == 0 && j == 0 && length == m->ns) {
    //     printf("%3.5e \n", d_lhs[(start + j) * m->nb_panels + i]);
    //     printf("%lu %lu %lu %lu \n", v0, v1, v2, v3);
        // printf("%.5e %.5e %.5e \n", inf.x, inf.y, inf.z);
        // printf("colloc %.4e %.4e %.4e \n", colloc.x, colloc.y, colloc.z);
        // printf("normal %.4e %.4e %.4e \n", normal.x, normal.y, normal.z);
    // }
}

__global__ void kernel_influence_cuda2(
    const MeshProxy* m,
    float* d_lhs,
    const uint64_t start, const uint64_t length, const uint64_t offset, const float sigma) {

    u64 j = blockIdx.y * blockDim.y + threadIdx.y;
    u64 i = blockIdx.x * blockDim.x + threadIdx.x;

    if (j >= length || i >= m->nb_panels) return;

    __shared__ float sharedVertexX[BlockSizeY * 4]; // v0 v1 v2 v3 v0 v1 v2 v3 v0 v1 v2 v3 v0 v1 v2 v3
    __shared__ float sharedVertexY[BlockSizeY * 4];
    __shared__ float sharedVertexZ[BlockSizeY * 4];
    __shared__ float sharedCollocX[BlockSizeX];
    __shared__ float sharedCollocY[BlockSizeX];
    __shared__ float sharedCollocZ[BlockSizeX];
    __shared__ float sharedNormalX[BlockSizeX];
    __shared__ float sharedNormalY[BlockSizeX];
    __shared__ float sharedNormalZ[BlockSizeX];

    if (threadIdx.x == 0) {
        const u64 v0 = (start + offset + j) + (start + offset + j) / m->ns;
        const u64 v1 = v0 + 1;
        const u64 v3 = v0 + m->ns + 1;
        const u64 v2 = v3 + 1;
        // Assuming each thread along the y dimension loads data for one 'vertex' and its reuses across x dimension.
        // Adjust the calculation of v0, v1, etc., based on the actual indices needed for each thread.
        int indexBase = threadIdx.y * 4; // Base index for storing data in shared memory, assuming 4 vertices needed per thread in y
        sharedVertexX[indexBase + 0] = m->v.x[v0];
        sharedVertexX[indexBase + 1] = m->v.x[v1];
        sharedVertexX[indexBase + 2] = m->v.x[v2];
        sharedVertexX[indexBase + 3] = m->v.x[v3];
        sharedVertexY[indexBase + 0] = m->v.y[v0];
        sharedVertexY[indexBase + 1] = m->v.y[v1];
        sharedVertexY[indexBase + 2] = m->v.y[v2];
        sharedVertexY[indexBase + 3] = m->v.y[v3];
        sharedVertexZ[indexBase + 0] = m->v.z[v0];
        sharedVertexZ[indexBase + 1] = m->v.z[v1];
        sharedVertexZ[indexBase + 2] = m->v.z[v2];
        sharedVertexZ[indexBase + 3] = m->v.z[v3];
    }

    // Example for loading colloc/normal data by threads along the y dimension
    if (threadIdx.y == 0) {
        // Load colloc and normal data into shared memory
        // Adjust indices based on your data structure and ensure no out-of-bounds access
        sharedCollocX[threadIdx.x] = m->colloc.x[i];
        sharedCollocY[threadIdx.x] = m->colloc.y[i];
        sharedCollocZ[threadIdx.x] = m->colloc.z[i];
        sharedNormalX[threadIdx.x] = m->normal.x[i];
        sharedNormalY[threadIdx.x] = m->normal.y[i];
        sharedNormalZ[threadIdx.x] = m->normal.z[i];
    }

    __syncthreads(); // Synchronize to ensure all data is loaded before proceeding

    const float3 vertex0{sharedVertexX[threadIdx.y * 4 + 0], sharedVertexY[threadIdx.y * 4 + 0], sharedVertexZ[threadIdx.y * 4 + 0]};
    const float3 vertex1{sharedVertexX[threadIdx.y * 4 + 1], sharedVertexY[threadIdx.y * 4 + 1], sharedVertexZ[threadIdx.y * 4 + 1]};
    const float3 vertex2{sharedVertexX[threadIdx.y * 4 + 2], sharedVertexY[threadIdx.y * 4 + 2], sharedVertexZ[threadIdx.y * 4 + 2]};
    const float3 vertex3{sharedVertexX[threadIdx.y * 4 + 3], sharedVertexY[threadIdx.y * 4 + 3], sharedVertexZ[threadIdx.y * 4 + 3]};

    const float3 colloc = {sharedCollocX[threadIdx.x], sharedCollocY[threadIdx.x], sharedCollocZ[threadIdx.x]};
    const float3 normal = {sharedNormalX[threadIdx.x], sharedNormalY[threadIdx.x], sharedNormalZ[threadIdx.x]};
    float3 inf{0.0f, 0.0f, 0.0f};
    kernel_symmetry(&inf, colloc, vertex0, vertex1, sigma);
    kernel_symmetry(&inf, colloc, vertex1, vertex2, sigma);
    kernel_symmetry(&inf, colloc, vertex2, vertex3, sigma);
    kernel_symmetry(&inf, colloc, vertex3, vertex0, sigma);
    d_lhs[(start + j) * m->nb_panels + i] += dot(inf, normal);
}

__global__ void kernel_influence_cuda3(
    const MeshProxy* m,
    float* d_lhs,
    const uint64_t start, const uint64_t length, const uint64_t offset, const float sigma) {

    u64 j = blockIdx.y * blockDim.y + threadIdx.y;
    u64 i = blockIdx.x * blockDim.x + threadIdx.x;

    if (j >= length || i >= m->nb_panels) return;

    __shared__ float sharedVertexX[BlockSizeY * 4]; // v0 v0 v0 v0 v1 v1 v1 v1 v2 v2 v2 v2 v3 v3 v3 v3
    __shared__ float sharedVertexY[BlockSizeY * 4];
    __shared__ float sharedVertexZ[BlockSizeY * 4];
    __shared__ float sharedCollocX[BlockSizeX];
    __shared__ float sharedCollocY[BlockSizeX];
    __shared__ float sharedCollocZ[BlockSizeX];
    __shared__ float sharedNormalX[BlockSizeX];
    __shared__ float sharedNormalY[BlockSizeX];
    __shared__ float sharedNormalZ[BlockSizeX];

    if (threadIdx.x == 0) {
        const u64 v0 = (start + offset + j) + (start + offset + j) / m->ns;
        const u64 v1 = v0 + 1;
        const u64 v3 = v0 + m->ns + 1;
        const u64 v2 = v3 + 1;
        // Assuming each thread along the y dimension loads data for one 'vertex' and its reuses across x dimension.
        // Adjust the calculation of v0, v1, etc., based on the actual indices needed for each thread.
        int indexBase = threadIdx.y; // Base index for storing data in shared memory, assuming 4 vertices needed per thread in y
        sharedVertexX[indexBase + 0 * BlockSizeY] = m->v.x[v0];
        sharedVertexX[indexBase + 1 * BlockSizeY] = m->v.x[v1];
        sharedVertexX[indexBase + 2 * BlockSizeY] = m->v.x[v2];
        sharedVertexX[indexBase + 3 * BlockSizeY] = m->v.x[v3];
        sharedVertexY[indexBase + 0 * BlockSizeY] = m->v.y[v0];
        sharedVertexY[indexBase + 1 * BlockSizeY] = m->v.y[v1];
        sharedVertexY[indexBase + 2 * BlockSizeY] = m->v.y[v2];
        sharedVertexY[indexBase + 3 * BlockSizeY] = m->v.y[v3];
        sharedVertexZ[indexBase + 0 * BlockSizeY] = m->v.z[v0];
        sharedVertexZ[indexBase + 1 * BlockSizeY] = m->v.z[v1];
        sharedVertexZ[indexBase + 2 * BlockSizeY] = m->v.z[v2];
        sharedVertexZ[indexBase + 3 * BlockSizeY] = m->v.z[v3];
    }

    // Example for loading colloc/normal data by threads along the y dimension
    if (threadIdx.y == 0) {
        // Load colloc and normal data into shared memory
        // Adjust indices based on your data structure and ensure no out-of-bounds access
        sharedCollocX[threadIdx.x] = m->colloc.x[i];
        sharedCollocY[threadIdx.x] = m->colloc.y[i];
        sharedCollocZ[threadIdx.x] = m->colloc.z[i];
        sharedNormalX[threadIdx.x] = m->normal.x[i];
        sharedNormalY[threadIdx.x] = m->normal.y[i];
        sharedNormalZ[threadIdx.x] = m->normal.z[i];
    }

    __syncthreads(); // Synchronize to ensure all data is loaded before proceeding

    const float3 vertex0{sharedVertexX[threadIdx.y + 0 * BlockSizeY], sharedVertexY[threadIdx.y + 0 * BlockSizeY], sharedVertexZ[threadIdx.y + 0 * BlockSizeY]};
    const float3 vertex1{sharedVertexX[threadIdx.y + 1 * BlockSizeY], sharedVertexY[threadIdx.y + 1 * BlockSizeY], sharedVertexZ[threadIdx.y + 1 * BlockSizeY]};
    const float3 vertex2{sharedVertexX[threadIdx.y + 2 * BlockSizeY], sharedVertexY[threadIdx.y + 2 * BlockSizeY], sharedVertexZ[threadIdx.y + 2 * BlockSizeY]};
    const float3 vertex3{sharedVertexX[threadIdx.y + 3 * BlockSizeY], sharedVertexY[threadIdx.y + 3 * BlockSizeY], sharedVertexZ[threadIdx.y + 3 * BlockSizeY]};

    const float3 colloc = {sharedCollocX[threadIdx.x], sharedCollocY[threadIdx.x], sharedCollocZ[threadIdx.x]};
    const float3 normal = {sharedNormalX[threadIdx.x], sharedNormalY[threadIdx.x], sharedNormalZ[threadIdx.x]};
    float3 inf{0.0f, 0.0f, 0.0f};
    kernel_symmetry(&inf, colloc, vertex0, vertex1, sigma);
    kernel_symmetry(&inf, colloc, vertex1, vertex2, sigma);
    kernel_symmetry(&inf, colloc, vertex2, vertex3, sigma);
    kernel_symmetry(&inf, colloc, vertex3, vertex0, sigma);
    d_lhs[(start + j) * m->nb_panels + i] += dot(inf, normal);
}

constexpr u64 get_grid_size(u64 length, u64 block_size) {
    return (length + block_size - 1) / block_size;
}

void BackendCUDA::compute_lhs(const FlowData& flow) {
    SimpleTimer timer("LHS");
    // Copy the latest mesh that has been corrected for the aoa
    u64 npt = mesh.nb_panels_total();
    u64 nvt = mesh.nb_vertices_total();
    CHECK_CUDA(hipMemcpy(h_mesh.v.x, mesh.v.x.data(), nvt * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(h_mesh.v.y, mesh.v.y.data(), nvt * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(h_mesh.v.z, mesh.v.z.data(), nvt * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(h_mesh.colloc.x, mesh.colloc.x.data(), npt * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(h_mesh.colloc.y, mesh.colloc.y.data(), npt * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(h_mesh.colloc.z, mesh.colloc.z.data(), npt * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(h_mesh.normal.x, mesh.normal.x.data(), npt * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(h_mesh.normal.y, mesh.normal.y.data(), npt * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(h_mesh.normal.z, mesh.normal.z.data(), npt * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_mesh, &h_mesh, sizeof(MeshProxy), hipMemcpyHostToDevice));
    CHECK_CUDA(hipDeviceSynchronize());

    dim3 block_size(BlockSizeX, BlockSizeY); // 1024 threads per block
    dim3 grid_size(get_grid_size(mesh.nb_panels_wing(), block_size.x), get_grid_size((mesh.nc - 1) * mesh.ns, block_size.y));
    kernel_influence_cuda<<<grid_size, block_size>>>(d_mesh, d_lhs, 0, (mesh.nc - 1) * mesh.ns, 0, flow.sigma_vatistas);
    
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error after kernel launch: %s\n", hipGetErrorString(error));
    }
    
    CHECK_CUDA(hipDeviceSynchronize());

    dim3 grid_size2(get_grid_size(mesh.nb_panels_wing(), block_size.x), get_grid_size(mesh.ns, block_size.y));
    for (u64 offset = 0; offset < mesh.nw + 1; offset++) {
        kernel_influence_cuda<<<grid_size2, block_size>>>(d_mesh, d_lhs, (mesh.nc - 1) * mesh.ns, mesh.ns, offset*mesh.ns, flow.sigma_vatistas);
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            fprintf(stderr, "CUDA Error after kernel launch: %s\n", hipGetErrorString(error));
        }
        CHECK_CUDA(hipDeviceSynchronize());
    }
}

void BackendCUDA::compute_rhs(const FlowData& flow) {
    default_backend.compute_rhs(flow);
}

void BackendCUDA::compute_rhs(const FlowData& flow, const std::vector<f32>& section_alphas) {
    default_backend.compute_rhs(flow, section_alphas);
}

int CUDA_LU_solver(hipsolverHandle_t handle, float *d_A,
                   float *d_b, int n) {
    // All pointers are device pointers
    // A is column major 
    // Ax = b -> result is stored in b
    int bufferSize = 0;
    int *info = NULL;
    float *buffer = NULL;
    int *ipiv = NULL;  // pivoting sequence
    int h_info = 0;

    CHECK_CUSOLVER(hipsolverDnSgetrf_bufferSize(handle, n, n, (float *)d_A,
                                                n, &bufferSize));

    CHECK_CUDA(hipMalloc((void**)&info, sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&buffer, sizeof(float) * bufferSize));
    CHECK_CUDA(hipMalloc((void**)&ipiv, sizeof(int) * n));

    CHECK_CUDA(hipMemset(info, 0, sizeof(int)));

    CHECK_CUSOLVER(hipsolverDnSgetrf(handle, n, n, d_A, n, buffer, ipiv, info));
    CHECK_CUDA(
        hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost));

    if (0 != h_info) {
    fprintf(stderr, "Error: LU factorization failed\n");
    }

    CHECK_CUSOLVER(
        hipsolverDnSgetrs(handle, HIPBLAS_OP_N, n, 1, d_A, n, ipiv, d_b, n, info));
    CHECK_CUDA(hipDeviceSynchronize());

    if (info) {
    CHECK_CUDA(hipFree(info));
    }
    if (buffer) {
    CHECK_CUDA(hipFree(buffer));
    }
    if (ipiv) {
    CHECK_CUDA(hipFree(ipiv));
    }

    return 0;
}

void BackendCUDA::lu_factor() {};

void BackendCUDA::lu_solve() {
    SimpleTimer timer("Solve");
    //default_backend.solve();
    auto& ctx = CtxManager::getInstance();
    u64 N = (u64)mesh.nb_panels_wing();

    // copy data to device
    //CHECK_CUDA(hipMemcpy(d_lhs, default_backend.lhs.data(), N*N * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_rhs, default_backend.rhs.data(), N * sizeof(float), hipMemcpyHostToDevice));
    
    // Solve on device
    CUDA_LU_solver(ctx.cusolver(), d_lhs, d_rhs, N);

    // copy data back to host
    CHECK_CUDA(hipMemcpy(default_backend.gamma.data(), d_rhs, N * sizeof(float), hipMemcpyDeviceToHost));
}

f32 BackendCUDA::compute_coefficient_cl(
    const FlowData& flow,
    const f32 area,
    const u64 j,
    const u64 n) {
    return default_backend.compute_coefficient_cl(flow, area, j, n);
}

f32 BackendCUDA::compute_coefficient_cd(
    const FlowData& flow,
    const f32 area,
    const u64 j,
    const u64 n) {
    return default_backend.compute_coefficient_cd(flow, area, j, n);
}

linalg::alias::float3 BackendCUDA::compute_coefficient_cm(
    const FlowData& flow,
    const f32 area,
    const f32 chord,
    const u64 j,
    const u64 n) {
    return default_backend.compute_coefficient_cm(flow, area, chord, j, n);
}

void BackendCUDA::compute_delta_gamma() {
    default_backend.compute_delta_gamma();
}
